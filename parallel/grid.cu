#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <iterator>
#include <omp.h>
#include <hip/hip_runtime.h>

#include "grid.h"
#include "utils.h"

Grid::Grid(int rows, int cols) : r(rows), c(cols){
    this->grid = new int[r * c];
    this->clear();
}

Grid::~Grid() {
    delete this->grid;
}

int Grid::rows() {return this->r;}
int Grid::cols() {return this->c;}

void Grid::clear() {
    for (int i = 0; i < this->r * this->c; ++i)
        grid[i] = 0;
}

void Grid::print() {
    using namespace std;
    for (int row = 0; row < this->r; ++row){
        for (int col = 0; col < this->c; ++col) {
            cout << (*this)(row, col) << "\t";
        }
        cout << endl;
    }
}

void Grid::populateFromFile(std::string filename) {
    using namespace std;

    int points = utils::getFileSize(filename) / sizeof(float) / 2;
    float * values = utils::readFile(filename);

    float bucket_row_width = 1.0f / this->r;
    float bucket_col_width = 1.0f / this->c;

    for (int p = 0; p < points; ++p){
        float x = values[2 * p];
        float y = values[2 * p + 1];

        int bucket_row = (int)(x / bucket_row_width);
        if (bucket_row >= this->r)
            bucket_row = this->r - 1;

        int bucket_col = (int)(y / bucket_col_width);
        if (y >= this->c)
            bucket_col = this->c - 1;

        ++this->grid[bucket_col + bucket_row * this->r];
    }

    delete values;
}

void Grid::populateFromArray(int points, float * values) {
    using namespace std;

    float bucket_row_width = 1.0f / this->r;
    float bucket_col_width = 1.0f / this->c;

    for (int p = 0; p < points; ++p){
        float x = values[2 * p];
        float y = values[2 * p + 1];

        int bucket_row = (int)(x / bucket_row_width);
        if (bucket_row >= this->r)
            bucket_row = this->r - 1;

        int bucket_col = (int)(y / bucket_col_width);
        if (y >= this->c)
            bucket_col = this->c - 1;

        ++this->grid[bucket_col + bucket_row * this->r];
    }
}

__device__ int partition(int * list, int left, int right, int pivot_index) {
    int pivot_value = list[pivot_index];

    // Move pivot to the end
    list[pivot_index] = list[right];
    list[right] = pivot_value;

    int store_index = left;

    for (int i = left; i < right; ++i) {
        if (list[i] < pivot_value) {
            int temp = list[store_index];
            list[store_index] = list[i];
            list[i] = temp;

            ++store_index;
        }
    }

    int temp = list[store_index];
    list[store_index] = list[right];
    list[right] = temp;

    return store_index;
}

__device__ int select_kth(int * list, int left, int right, int k) {
    if (left == right)
        return list[left];

    int pivot_index;
    while(true) {
        pivot_index = left + (right - left) / 2;// + (int)(((float)rand() / RAND_MAX) * (right - left));
        pivot_index = partition(list, left, right, pivot_index);

        if (k == pivot_index)
            return list[k];
        else if (k < pivot_index)
            right = pivot_index - 1;
        else
            left = pivot_index + 1;
    }
}

__device__ int clamp(int x, int a, int b) {
    return max(a, min(b, x));
}

__global__ void filter(int * grid, int * result, int rows, int cols, int diameter) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row < rows and col < cols){
        int top = clamp(row - (diameter - 1) / 2, 0, rows - 1);
        int bottom = clamp(row + (diameter - 1) / 2, 0, rows - 1);
        int left = clamp(col - (diameter - 1) / 2, 0, cols - 1);
        int right = clamp(col + (diameter - 1) / 2, 0, cols - 1);

        int num_values = (bottom - top + 1) * (right - left + 1);
        int values[441];
        
        int count = 0;

        for (int r = top; r <= bottom; ++r) {
            for (int c = left; c <= right; ++c) {
                values[count] = grid[c + r * cols];
                ++count;
            }
        }

        result[col + row * cols] = select_kth(values, 0, num_values - 1, (num_values - 1) / 2);
    }
}

void CUDA_SAFE_CALL(hipError_t error, std::string error_message) {
    using namespace std;
    if(error!=hipSuccess)
    {
       cout << error_message << endl;
       cout << hipGetErrorString(error) << endl;
       exit(-1);
    }
}

void Grid::applyMedianFilter(int diameter) {
    using namespace std;

    int * new_grid = new int[this->r * this->c];

    CUDA_SAFE_CALL(hipSetDevice(0), "Failed to set device");
    CUDA_SAFE_CALL(
        hipMalloc(&d_grid, this->r * this->c * sizeof(float)),
        "Failed to allocate memory for d_grid"
    );
    CUDA_SAFE_CALL(
        hipMalloc(&d_result, this->r * this->c * sizeof(float)),
        "Failed to allocate memory for d_result"
    );
    
    CUDA_SAFE_CALL(
        hipMemcpy(d_grid, this->grid, this->r * this->c * sizeof(float), hipMemcpyHostToDevice),
        "Failed to copy grid to d_grid"
    );

    dim3 dimBlock(16,16);
    dim3 dimGrid(
        this->r / dimBlock.x + 1,
        this->c / dimBlock.y + 1
    );

    filter<<<dimGrid, dimBlock>>>(d_grid, d_result, this->r, this->c, diameter);

    CUDA_SAFE_CALL(hipDeviceSynchronize(), "Running kernel failed");

    CUDA_SAFE_CALL(hipMemcpy(
        new_grid, d_result, this->r * this->c * sizeof(float), hipMemcpyDeviceToHost),
        "Failed to copy d_result to new_grid"
    );

    CUDA_SAFE_CALL(hipFree(d_grid), "Failed to free d_grid");
    CUDA_SAFE_CALL(hipFree(d_result), "Failed to free d_result");

    delete [] grid;
    grid = new_grid;
}

void Grid::printToFile(std::string filename) {
    using namespace std;
    utils::outputToCSV(this->grid, this->r, this->c, filename);
}