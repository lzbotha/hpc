#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <iterator>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <cstdio>

#include "grid.h"
#include "utils.h"

Grid::Grid(int rows, int cols) : r(rows), c(cols){
    this->grid = new int[r * c]();
    this->clear();
}

Grid::~Grid() {
    delete this->grid;
}

int Grid::rows() {return this->r;}
int Grid::cols() {return this->c;}

void Grid::clear() {
    for (int i = 0; i < this->r * this->c; ++i)
        grid[i] = 0;
}

void Grid::print() {
    using namespace std;
    for (int row = 0; row < this->r; ++row){
        for (int col = 0; col < this->c; ++col) {
            cout << (*this)(row, col) << "\t";
        }
        cout << endl;
    }
}

void Grid::populateFromFile(std::string filename) {
    using namespace std;

    int points = utils::getFileSize(filename) / sizeof(float) / 2;
    float * values = utils::readFile(filename);

    float bucket_row_width = 1.0f / this->r;
    float bucket_col_width = 1.0f / this->c;

    for (int p = 0; p < points; ++p){
        float x = values[2 * p];
        float y = values[2 * p + 1];

        int bucket_row = (int)(x / bucket_row_width);
        if (bucket_row >= this->r)
            bucket_row = this->r - 1;

        int bucket_col = (int)(y / bucket_col_width);
        if (y >= this->c)
            bucket_col = this->c - 1;

        ++this->grid[bucket_col + bucket_row * this->r];
    }

    delete values;
}

void Grid::populateFromArray(int points, float * values) {
    using namespace std;

    float bucket_row_width = 1.0f / this->r;
    float bucket_col_width = 1.0f / this->c;

    for (int p = 0; p < points; ++p){
        float x = values[2 * p];
        float y = values[2 * p + 1];

        int bucket_row = (int)(x / bucket_row_width);
        if (bucket_row >= this->r)
            bucket_row = this->r - 1;

        int bucket_col = (int)(y / bucket_col_width);
        if (y >= this->c)
            bucket_col = this->c - 1;

        ++this->grid[bucket_col + bucket_row * this->r];
    }
}

// void Grid::populateFromArray(int points, float * values) {
//     using namespace std;

//     float bucket_row_width = 1.0f / this->r;
//     float bucket_col_width = 1.0f / this->c;

//     int num_threads = omp_get_num_procs();
//     int ** grids = new int*[num_threads];

//     int current = 0;
//     omp_lock_t current_lock;
//     omp_init_lock(&current_lock);

//     #pragma omp parallel
//     {
//         int tid = omp_get_thread_num();
//         grids[tid] = new int[this->r * this->c]();
//     }

//     // #pragma omp parallel for
//     for (int p = 0; p < points; ++p){
//         int tid = omp_get_thread_num();

//         float x;
//         float y;

//         omp_set_lock(&current_lock);

//         if (current < points) {
//             x = values[2 * current];
//             y = values[2 * current + 1];

//             ++current;
//             omp_unset_lock(&current_lock);
//         }
//         else {
//             omp_unset_lock(&current_lock);
//             continue;
//         }
                

//         int bucket_row = (int)(x / bucket_row_width);
//         if (bucket_row >= this->r)
//             bucket_row = this->r - 1;

//         int bucket_col = (int)(y / bucket_col_width);
//         if (y >= this->c)
//             bucket_col = this->c - 1;

//         ++grids[tid][bucket_col + bucket_row * this->c];
//     }

//     omp_destroy_lock(&current_lock);


//     int row_offset = this->r / num_threads + 1;
//     #pragma omp parallel
//     {
//         int tid = omp_get_thread_num();

//         int current = tid * row_offset;
//         int end = current + row_offset;

//         while (current < end and current < this->r) {

//             for (int c = 0; c < this->c; ++c)
//                 for (int g = 0; g < num_threads; ++g)
//                     this->grid[c + current * this->c] += grids[g][c + current * this->c];

//             ++current;
//         }
//     }

//     for (int i = 0; i < num_threads; ++i)
//         delete[] grids[i];
//     delete[] grids;
// }

// void Grid::populateFromArray(int points, float * values) {
//     using namespace std;

//     float bucket_row_width = 1.0f / this->r;
//     float bucket_col_width = 1.0f / this->c;

//     int num_threads = omp_get_num_procs();
//     int ** grids = new int*[num_threads];
//     int offset = points / num_threads + 1;
//     if (offset % 2 == 1)
//         ++offset;
    
//     #pragma omp parallel
//     {
//         int tid = omp_get_thread_num();
//         grids[tid] = new int[this->r * this->c]();

//         int current = tid * offset;
//         int end = current + offset;

//         while (current < end and current < points) {
//             float x = values[2 * current];
//             float y = values[2 * current + 1];

//             int bucket_row = (int)(x / bucket_row_width);
//             if (bucket_row >= this->r)
//                 bucket_row = this->r - 1;

//             int bucket_col = (int)(y / bucket_col_width);
//             if (y >= this->c)
//                 bucket_col = this->c - 1;

//             ++grids[tid][bucket_col + bucket_row * this->c];
//             ++current;
//         }
//     }

//     int row_offset = this->r / num_threads + 1;
//     #pragma omp parallel
//     {
//         int tid = omp_get_thread_num();

//         int current = tid * row_offset;
//         int end = current + row_offset;

//         while (current < end and current < this->r) {

//             for (int c = 0; c < this->c; ++c)
//                 for (int g = 0; g < num_threads; ++g)
//                     this->grid[c + current * this->c] += grids[g][c + current * this->c];

//             ++current;
//         }
//     }



//     for (int i = 0; i < num_threads; ++i)
//         delete[] grids[i];
//     delete[] grids;
// }

__device__ int partition(int * list, int left, int right, int pivot_index) {
    int pivot_value = list[pivot_index];

    // Move pivot to the end
    list[pivot_index] = list[right];
    list[right] = pivot_value;

    int store_index = left;

    for (int i = left; i < right; ++i) {
        if (list[i] < pivot_value) {
            int temp = list[store_index];
            list[store_index] = list[i];
            list[i] = temp;

            ++store_index;
        }
    }

    int temp = list[store_index];
    list[store_index] = list[right];
    list[right] = temp;

    return store_index;
}

__device__ int select_kth(int * list, int left, int right, int k) {
    if (left == right)
        return list[left];

    int pivot_index;
    while(true) {
        pivot_index = left + (right - left) / 2;// + (int)(((float)rand() / RAND_MAX) * (right - left));
        pivot_index = partition(list, left, right, pivot_index);

        if (k == pivot_index)
            return list[k];
        else if (k < pivot_index)
            right = pivot_index - 1;
        else
            left = pivot_index + 1;
    }
}

__device__ int clamp(int x, int a, int b) {
    return max(a, min(b, x));
}

// __global__ void filter(int * grid, int * result, int rows, int cols, int diameter) {
//     int row = blockIdx.x * blockDim.x + threadIdx.x;
//     int col = blockIdx.y * blockDim.y + threadIdx.y;

//     if(row < rows and col < cols){
//         int top = clamp(row - (diameter - 1) / 2, 0, rows - 1);
//         int bottom = clamp(row + (diameter - 1) / 2, 0, rows - 1);
//         int left = clamp(col - (diameter - 1) / 2, 0, cols - 1);
//         int right = clamp(col + (diameter - 1) / 2, 0, cols - 1);

//         int num_values = (bottom - top + 1) * (right - left + 1);
//         int values[441];
        
//         int count = 0;

//         for (int r = top; r <= bottom; ++r) {
//             for (int c = left; c <= right; ++c) {
//                 values[count] = grid[c + r * cols];
//                 ++count;
//             }
//         }

//         result[col + row * cols] = select_kth(values, 0, num_values - 1, (num_values - 1) / 2);
//     }
// }

__global__ void filter(int * grid, int * result, int rows, int cols, int diameter) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ int cache[1296];
    int tlcx = blockIdx.x * blockDim.x - 10;
    int tlcy = blockIdx.y * blockDim.y - 10;

    // int index = (threadIdx.y + threadIdx.x * blockDim.y) * 5;
    // for (int i = 0; i < 5;++i){
    //     int current_index = ((index + i) % 36 + tlcy) + ((index + i) / 36 + tlcx) * 36;

    //     if (current_index < rows and current_index > 0)
    //         cache[index + i] = grid[current_index];
    // }

    if (threadIdx.x == 0 and threadIdx.y == 0) {
        for (int r = 0; r < 36; ++r){
            for (int c = 0; c < 36; ++c) {
                if (c + tlcy >= 0 and r + tlcx >= 0 and c + tlcy < cols and r + tlcx < rows)
                    cache[c + r * 36] = grid[(c + tlcy) + (r + tlcx) * rows];
            }
        }
    }

    __syncthreads();

    if(row < rows and col < cols){
        int values[441];
        int count = 0;

        int r_start = 0;
        int r_end = 21;
        int c_start = 0;
        int c_end = 21;

        if (blockIdx.x == 0)
            r_start = 10;
        if (blockIdx.x == gridDim.x)
            r_end = 11;
        if (blockIdx.y == 0)
            c_start = 10;
        if (blockIdx.y == gridDim.y)
            r_end = 11;

        for (int r = r_start; r < r_end; ++r) {
            for (int c = c_start; c < c_end; ++c) {
                values[count] = cache[(c + threadIdx.y) + (r + threadIdx.x) * 36];
                ++count;
            }
        }

        result[col + row * cols] = select_kth(values, 0, count - 1, (count - 1) / 2);
    }
}

void CUDA_SAFE_CALL(hipError_t error, std::string error_message) {
    using namespace std;
    if(error!=hipSuccess)
    {
       cout << error_message << endl;
       cout << hipGetErrorString(error) << endl;
       exit(-1);
    }
}

void Grid::applyMedianFilter(int diameter) {
    using namespace std;

    int * new_grid = new int[this->r * this->c];

    CUDA_SAFE_CALL(hipSetDevice(0), "Failed to set device");
    CUDA_SAFE_CALL(
        hipMalloc(&d_grid, this->r * this->c * sizeof(float)),
        "Failed to allocate memory for d_grid"
    );
    CUDA_SAFE_CALL(
        hipMalloc(&d_result, this->r * this->c * sizeof(float)),
        "Failed to allocate memory for d_result"
    );
    
    CUDA_SAFE_CALL(
        hipMemcpy(d_grid, this->grid, this->r * this->c * sizeof(float), hipMemcpyHostToDevice),
        "Failed to copy grid to d_grid"
    );

    dim3 dimBlock(16,16);
    dim3 dimGrid(
        this->r / dimBlock.x + 1,
        this->c / dimBlock.y + 1
    );

    filter<<<dimGrid, dimBlock>>>(d_grid, d_result, this->r, this->c, diameter);

    CUDA_SAFE_CALL(hipDeviceSynchronize(), "Running kernel failed");

    CUDA_SAFE_CALL(hipMemcpy(
        new_grid, d_result, this->r * this->c * sizeof(float), hipMemcpyDeviceToHost),
        "Failed to copy d_result to new_grid"
    );

    CUDA_SAFE_CALL(hipFree(d_grid), "Failed to free d_grid");
    CUDA_SAFE_CALL(hipFree(d_result), "Failed to free d_result");

    delete [] grid;
    grid = new_grid;
}

void Grid::printToFile(std::string filename) {
    using namespace std;
    utils::outputToCSV(this->grid, this->r, this->c, filename);
}