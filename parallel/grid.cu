#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <iterator>
#include <omp.h>
#include <hip/hip_runtime.h>

#include "grid.h"
#include "utils.h"

Grid::Grid(int rows, int cols) : r(rows), c(cols){
    this->grid = new int[r * c];
    this->clear();
}

Grid::~Grid() {
    delete this->grid;
}

int Grid::rows() {return this->r;}
int Grid::cols() {return this->c;}

void Grid::clear() {
    for (int i = 0; i < this->r * this->c; ++i)
        grid[i] = 0;
}

void Grid::print() {
    using namespace std;
    for (int row = 0; row < this->r; ++row){
        for (int col = 0; col < this->c; ++col) {
            cout << (*this)(row, col) << "\t";
        }
        cout << endl;
    }
}

void Grid::populateFromFile(std::string filename) {
    using namespace std;

    int points = utils::getFileSize(filename) / sizeof(float) / 2;
    float * values = utils::readFile(filename);

    float bucket_row_width = 1.0f / this->r;
    float bucket_col_width = 1.0f / this->c;

    for (int p = 0; p < points; ++p){
        float x = values[2 * p];
        float y = values[2 * p + 1];

        int bucket_row = (int)(x / bucket_row_width);
        if (bucket_row >= this->r)
            bucket_row = this->r - 1;

        int bucket_col = (int)(y / bucket_col_width);
        if (y >= this->c)
            bucket_col = this->c - 1;

        ++this->grid[bucket_col + bucket_row * this->r];
    }

    delete values;
}

void Grid::populateFromArray(int points, float * values) {
    using namespace std;

    float bucket_row_width = 1.0f / this->r;
    float bucket_col_width = 1.0f / this->c;

    for (int p = 0; p < points; ++p){
        float x = values[2 * p];
        float y = values[2 * p + 1];

        int bucket_row = (int)(x / bucket_row_width);
        if (bucket_row >= this->r)
            bucket_row = this->r - 1;

        int bucket_col = (int)(y / bucket_col_width);
        if (y >= this->c)
            bucket_col = this->c - 1;

        ++this->grid[bucket_col + bucket_row * this->r];
    }
}

__device__ int partition(int * list, int left, int right, int pivot_index) {
    int pivot_value = list[pivot_index];

    // Move pivot to the end
    list[pivot_index] = list[right];
    list[right] = pivot_value;

    int store_index = left;

    for (int i = left; i < right; ++i) {
        if (list[i] < pivot_value) {
            int temp = list[store_index];
            list[store_index] = list[i];
            list[i] = temp;

            ++store_index;
        }
    }

    int temp = list[store_index];
    list[store_index] = list[right];
    list[right] = temp;

    return store_index;
}

__device__ int select_kth(int * list, int left, int right, int k) {
    if (left == right)
        return list[left];

    int pivot_index;
    while(true) {
        pivot_index = left + (right - left) / 2;// + (int)(((float)rand() / RAND_MAX) * (right - left));
        pivot_index = partition(list, left, right, pivot_index);

        if (k == pivot_index)
            return list[k];
        else if (k < pivot_index)
            right = pivot_index - 1;
        else
            left = pivot_index + 1;
    }
}

__device__ int clamp(int x, int a, int b) {
    return max(a, min(b, x));
}

__global__ void filter(int * grid, int * result, int rows, int cols, int diameter) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row < rows and col < cols){
        int top = clamp(row - (diameter - 1) / 2, 0, rows - 1);
        int bottom = clamp(row + (diameter - 1) / 2, 0, rows - 1);
        int left = clamp(col - (diameter - 1) / 2, 0, cols - 1);
        int right = clamp(col + (diameter - 1) / 2, 0, cols - 1);

        int num_values = (bottom - top + 1) * (right - left + 1);
        int values[9];
        // int padding = 9 - diameter * diameter;

        // for (int i = 0; i < padding; ++i)
        //     values[i] = -1;

        // int count = padding;
        int count = 0;

        for (int r = top; r <= bottom; ++r) {
            for (int c = left; c <= right; ++c) {
                values[count] = grid[c + r * cols];
                ++count;
            }
        }

        result[col + row * cols] = select_kth(values, 0, num_values - 1, (num_values - 1) / 2);
        // result[col + row * cols] = padding + (num_values - 1) / 2 - 1;
    }
}


// int Grid::medianFilter(int row, int col, int diameter) {
//     using namespace std;

//     int top = utils::clamp(row - (diameter - 1) / 2, 0, this->r - 1);
//     int bottom = utils::clamp(row + (diameter - 1) / 2, 0, this->r - 1);
//     int left = utils::clamp(col - (diameter - 1) / 2, 0, this->c - 1);
//     int right = utils::clamp(col + (diameter - 1) / 2, 0, this->c - 1);

//     int num_values = (bottom - top + 1) * (right - left + 1);
//     int values[num_values];
//     int count = 0;

//     for (int r = top; r <= bottom; ++r) {
//         copy(
//             grid + left + r * this->r,
//             grid + right + r * this->r + 1,
//             values + count
//         );
//         count += right - left + 1;
//     }

//     int middle = (num_values - 1) / 2;

//     return select_kth(values, 0, num_values - 1, middle);
// }

void Grid::applyMedianFilter(int diameter) {
    using namespace std;

    int * new_grid = new int[this->r * this->c];

    hipSetDevice(0);
    hipMalloc(&d_grid, this->r * this->c * sizeof(float));
    hipMalloc(&d_result, this->r * this->c * sizeof(float));
    
    hipMemcpy(d_grid, this->grid, this->r * this->c * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(16,16);
    dim3 dimGrid(
        this->r / dimBlock.x + 1,
        this->c / dimBlock.y + 1
    );

    filter<<<dimGrid, dimBlock>>>(d_grid, d_result, this->r, this->c, diameter);
    
    hipMemcpy(new_grid, d_result, this->r * this->c * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_grid);
    hipFree(d_result);

    delete [] grid;
    grid = new_grid;

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
       cout << hipGetErrorString(error) << endl;
       exit(-1);
    }
}

void Grid::printToFile(std::string filename) {
    using namespace std;
    utils::outputToCSV(this->grid, this->r, this->c, filename);
}

// int Grid::select_kth(int * list, int left, int right, int k) {
//     if (left == right)
//         return list[left];

//     int pivot_index;
//     while(true) {
//         pivot_index = left + (int)(((float)rand() / RAND_MAX) * (right - left));
//         pivot_index = partition(list, left, right, pivot_index);

//         if (k == pivot_index)
//             return list[k];
//         else if (k < pivot_index)
//             right = pivot_index - 1;
//         else
//             left = pivot_index + 1;
//     }
// }

// inline int Grid::partition(int * list, int left, int right, int pivot_index) {
//     int pivot_value = list[pivot_index];

//     // Move pivot to the end
//     list[pivot_index] = list[right];
//     list[right] = pivot_value;

//     int store_index = left;

//     for (int i = left; i < right; ++i) {
//         if (list[i] < pivot_value) {
//             int temp = list[store_index];
//             list[store_index] = list[i];
//             list[i] = temp;

//             ++store_index;
//         }
//     }

//     int temp = list[store_index];
//     list[store_index] = list[right];
//     list[right] = temp;

//     return store_index;
// }